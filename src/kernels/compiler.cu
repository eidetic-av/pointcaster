#include "hip/hip_runtime.h"
#include "./compiler.h"
#include "../logger.h"
#include <hip/hip_runtime.h>
#include <iostream>
#include <hip/hiprtc.h>
#include <thrust/device_vector.h>
#include <thrust/for_each.h>

#define NVRTC_SAFE_CALL(Name, x)                                               \
  do {                                                                         \
    hiprtcResult result = x;                                                    \
    if (result != HIPRTC_SUCCESS) {                                             \
      std::cerr << "\nerror: " << Name << " failed with error "                \
		<< hiprtcGetErrorString(result);                                \
      exit(1);                                                                 \
    }                                                                          \
  } while (0)

__device__ void function(int x) { }

struct functor {

  __device__ void operator()(int x) {
    function(x);
  }
};

void for_each() {
  pc::logger->debug("for_each");
  thrust::device_vector<int> d_vec(3);
  d_vec[0] = 0;
  d_vec[0] = 1;
  d_vec[0] = 2;
  thrust::for_each(d_vec.begin(), d_vec.end(), functor());
}

std::string function_source = R"(

__device__ void function(int x) {
  printf("%d\n", x);
}

)";

namespace pc::kernels {

KernelCompiler::KernelCompiler() {
  pc::logger->info("Loading CUDA Kernel compiler");

  // Initialize CUDA context
  hipInit(0);
  hipDevice_t cuDevice;
  hipDeviceGet(&cuDevice, 0);
  hipCtx_t cuContext;
  hipCtxCreate(&cuContext, 0, cuDevice);

  // Define program to compile & run
  hiprtcProgram program;
  NVRTC_SAFE_CALL("create_program", hiprtcCreateProgram(&program, function_source.c_str(), "function", 0, nullptr, nullptr));

  // compile the program
  const char *options[] = {"-rdc=true"}; 
  hiprtcResult compile_result = hiprtcCompileProgram(program, 1, options);

  // obtain compilation log
  std::size_t log_size;
  NVRTC_SAFE_CALL("get_log_size", hiprtcGetProgramLogSize(program, &log_size));
  if (log_size > 1) {
    char* log;
    NVRTC_SAFE_CALL("get_log", hiprtcGetProgramLog(program, &log[0]));
    std::cout << &log[0] << "\n";
  }

  if (compile_result != HIPRTC_SUCCESS) exit(1);

  // obtain PTX from the program
  std::size_t ptx_size;
  NVRTC_SAFE_CALL("get_ptx_size", hiprtcGetCodeSize(program, &ptx_size));
  char *ptx = new char[ptx_size];
  NVRTC_SAFE_CALL("get_ptx", hiprtcGetCode(program, ptx));

  // Destroy the program
  NVRTC_SAFE_CALL("destroy_program", hiprtcDestroyProgram(&program));

  // Load precompiled relocatable source with call to external function and link
  // it together with NVRTC-compiled function

  hiprtcLinkState linker;
  CUDA_SAFE_CALL(hiprtcLinkCreate(0, NULL, NULL, &linker));
  CUDA_SAFE_CALL(
      hiprtcLinkAddFile(linker, hipJitInputPtx, "functor.ptx", 0, NULL, NULL));
  CUDA_SAFE_CALL(hiprtcLinkAddData(linker, hipJitInputPtx, (void *)ptx, ptx_size,
			       "function.ptx", 0, NULL, NULL));
  void *cubin;
  CUDA_SAFE_CALL(hiprtcLinkComplete(linker, &cubin, NULL));
  CUDA_SAFE_CALL(hipModuleLoadDataEx(&module, cubin, 0, NULL, NULL));
  CUDA_SAFE_CALL(hiprtcLinkDestroy(linker));

}

} // namespace pc::kernels
